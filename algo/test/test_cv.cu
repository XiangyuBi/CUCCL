#include "../CUCCL_LE/CUCCL_LE.hpp"
#include "../CUCCL_LE/CUCCL_LE.cuh"
#include "../CUCCL_NP/CUCCL_NP.cuh"
#include "../CUCCL_DPL/CUCCL_DPL.cuh"


#include <opencv2/core/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui/highgui.hpp>


#include <iomanip>
#include <iostream>

using namespace std; 
using namespace CUCCL; 
using namespace cv;
/*

void testCCL(char const* flag)
{
    const auto width = 32;
	const auto height = 8;
	unsigned char data[width * height] =
	{
		135, 135, 240, 240, 240, 135, 135, 135, 135, 135, 135, 135, 135, 135, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 135, 135, 135, 135, 135, 120, 120,
		135, 135, 240, 240, 240, 135, 135, 135, 135, 135, 135, 135, 135, 135, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 135, 135, 135, 135, 135, 120, 120,
		135, 135, 135, 135, 135, 135, 135, 135, 135, 135, 135, 135, 135, 135, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 135, 135, 135, 135, 120, 120,
		135, 135, 135, 135, 135, 135, 135, 135, 135, 135, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 135, 135, 135, 120, 120, 120,
		135, 135, 135, 135, 135, 135, 135, 135, 135, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120,
		135, 135, 135, 135, 135, 135, 135, 135, 135, 135, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120,
		135, 135, 135, 135, 135, 135, 135, 135, 135, 135, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120,
		135, 135, 135, 135, 135, 135, 135, 135, 135, 135, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120
    };
    
    int labels[width * height] = { 0 };

    cout << "Binary image is : " <<endl;
	for (auto i = 0; i < height; i++)
	{
		for (auto j = 0; j < width; j++)
		{
			cout << setw(3) << static_cast<int>(data[i * width + j]) << " ";
		}
		cout << endl;
	}
    cout<<endl;
    
    auto degreeOfConnectivity = 4;
    unsigned char threshold = 0;
    
    if (flag == "LE")
    {
        CCLLEGPU ccl;

        ccl.CudaCCL(data, labels, width, height, degreeOfConnectivity, threshold) ;
        cout << "Label Mesh by CCL LE : " <<endl;
	    for (auto i = 0; i < height; i++)
	    {
		    for (auto j = 0; j < width; j++)
		    {
			    cout << setw(3) << labels[i * width + j] << " ";
		    }
		    cout << endl;
        }
    }

    if (flag == "NP")
    {
        CCLNPGPU cclnp;
	    cclnp.CudaCCL(data, labels, width, height, degreeOfConnectivity, threshold);

	    cout << "Label Mesh by CCL NP : " << endl;
	    for (auto i = 0; i < height; i++)
	    {
		    for (auto j = 0; j < width; j++)
		    {
			    cout << setw(3) << labels[i * width + j] << " ";
		    }
		    cout << endl;
	    }
	}
	
	if (flag == "DPL")
	{
		CCLDPLGPU ccldpl;
		ccldpl.CudaCCL(data, labels, width, height, degreeOfConnectivity, threshold);

		cout << "Label Mesh by CCL DPL : " << endl;
		for (auto i = 0; i < height; i++)
		{
			for (auto j = 0; j < width; j++)
			{
				cout << setw(3) << labels[i * width + j] << " ";
			}
			cout << endl;
		}

	}
}

*/

void test_image( char const* path, char const* flag)
{
	Mat* image ;
	image = new Mat(imread(path, IMREAD_GRAYSCALE));
	int* label = new int[image->size().width * image->size().height]{0} ;
	unsigned char* data  = image->data ;
	auto degreeOfConnectivity = 4 ;
	unsigned char threshold = 0 ;
	if (flag == "DPL")
	{
		CCLDPLGPU ccldpl;
		ccldpl.CudaCCL(data, label, image->size().height, image->size().width, degreeOfConnectivity, threshold);
	}
	namedWindow( "Display window", WINDOW_AUTOSIZE ); // Create a window for display.
	Mat* result = new Mat(image->size().height, image->size().width , CV_32SC1, (void*)label );
	imshow( "Display window", *result );                // Show our image inside it.
	
	waitKey(0); // Wait for a keystroke in the window
	delete image ;
	delete label ;
	delete result ;
	image = nullptr ;
	label = nullptr ;
	result = nullptr ;
}
int main()
{
    test_image("test.png", "DPL");
	return 1 ;

}
