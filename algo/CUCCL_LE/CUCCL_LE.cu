#include "hip/hip_runtime.h"
#ifndef CUCCL_LE_CU
#define CUCCL_LE_CU

#include <cmath>
#include <>
#include <iostream>
#include <iomanip>
#include "CUCCL_LE.cuh"

namespace CUCCL{

const int BLOCK = 8;

__device__ int IMin(int a, int b)
{
	return a < b ? a : b;
}

__device__ unsigned char diff(unsigned char a, unsigned char b)
{
	return abs(a - b);
}

__global__ void InitCCL(int labelList[], int reference[], int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int id = x + y * width;

	labelList[id] = reference[id] = id;
}

__global__ void Scanning(unsigned char frame[], int labelList[], int reference[], bool* markFlag, int N, int width, int height, unsigned char threshold)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int id = x + y * width;

	unsigned char value = frame[id];
	int label = N;

	if (id - width >= 0 && diff(value, frame[id - width]) <= threshold)
		label = IMin(label, labelList[id - width]);
	if (id + width < N  && diff(value, frame[id + width]) <= threshold)
		label = IMin(label, labelList[id + width]);

	int col = id % width;

	if (col > 0 && diff(value, frame[id - 1]) <= threshold)
		label = IMin(label, labelList[id - 1]);
	if (col + 1 < width  && diff(value, frame[id + 1]) <= threshold)
		label = IMin(label, labelList[id + 1]);

	if (label < labelList[id])
	{
		reference[labelList[id]] = label;
		*markFlag = true;
	}
}

__global__ void scanning8(unsigned char frame[], int labelList[], int reference[], bool* markFlag, int N, int width, int height, unsigned char threshold)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int id = x + y * blockDim.x * gridDim.x;

	if (id >= N)
		return;

	unsigned char value = frame[id];
	int label = N;

	if (id - width >= 0 && diff(value, frame[id - width]) <= threshold)
		label = IMin(label, labelList[id - width]);

	if (id + width < N  && diff(value, frame[id + width]) <= threshold)
		label = IMin(label, labelList[id + width]);

	int col = id % width;
	if (col > 0)
	{
		if (diff(value, frame[id - 1]) <= threshold)
			label = IMin(label, labelList[id - 1]);
		if (id - width - 1 >= 0 && diff(value, frame[id - width - 1]) <= threshold)
			label = IMin(label, labelList[id - width - 1]);
		if (id + width - 1 < N  && diff(value, frame[id + width - 1]) <= threshold)
			label = IMin(label, labelList[id + width - 1]);
	}
	if (col + 1 < width)
	{
		if (diff(value, frame[id + 1]) <= threshold)
			label = IMin(label, labelList[id + 1]);
		if (id - width + 1 >= 0 && diff(value, frame[id - width + 1]) <= threshold)
			label = IMin(label, labelList[id - width + 1]);
		if (id + width + 1 < N  && diff(value, frame[id + width + 1]) <= threshold)
			label = IMin(label, labelList[id + width + 1]);
	}

	if (label < labelList[id])
	{
		reference[labelList[id]] = label;
		*markFlag = true;
	}
}

__global__ void analysis(int labelList[], int reference[], int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int id = x + y * width;

	int label = labelList[id];
	int ref;
	if (label == id)
	{
		do
		{
			ref = label;
			label = reference[ref];
		} while (ref ^ label);
		reference[id] = label;
	}
}

__global__ void labeling(int labelList[], int reference[], int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int id = x + y * width;

	labelList[id] = reference[reference[labelList[id]]];
}



void CCLLEGPU::CudaCCL(unsigned char* frame, int* labels, int width, int height, int degreeOfConnectivity, unsigned char threshold)
{
	auto N = width * height;

	hipMalloc(reinterpret_cast<void**>(&LabelListOnDevice), sizeof(int) * N);
	hipMalloc(reinterpret_cast<void**>(&ReferenceOnDevice), sizeof(int) * N);
	hipMalloc(reinterpret_cast<void**>(&FrameDataOnDevice), sizeof(unsigned char) * N);

	hipMemcpy(FrameDataOnDevice, frame, sizeof(unsigned char) * N, hipMemcpyHostToDevice);

	bool* markFlagOnDevice;
	hipMalloc(reinterpret_cast<void**>(&markFlagOnDevice), sizeof(bool));

	dim3 grid((width + BLOCK - 1) / BLOCK, (height + BLOCK - 1) / BLOCK);
	dim3 threads(BLOCK, BLOCK);

	InitCCL <<<grid, threads >>>(LabelListOnDevice, ReferenceOnDevice, width, height);

	auto initLabel = reinterpret_cast<int*>(malloc(sizeof(int) * width * height));

	hipMemcpy(initLabel, LabelListOnDevice, sizeof(int) * width * height, hipMemcpyDeviceToHost);
	std::cout << "Init labels:" << std::endl;
	for (auto i = 0; i < height; ++i)
	{
		for (auto j = 0; j < width; ++j)
		{
			std::cout << std::setw(3) << initLabel[i * width + j] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
	free(initLabel);

	while (true)
	{
		auto markFalgOnHost = false;
		hipMemcpy(markFlagOnDevice, &markFalgOnHost, sizeof(bool), hipMemcpyHostToDevice);

		if (degreeOfConnectivity == 4)
		{
			Scanning <<< grid, threads >>>(FrameDataOnDevice, LabelListOnDevice, ReferenceOnDevice, markFlagOnDevice, N, width, height, threshold);
			hipDeviceSynchronize();
		}
		else
			scanning8 <<< grid, threads >>>(FrameDataOnDevice, LabelListOnDevice, ReferenceOnDevice, markFlagOnDevice, N, width, height, threshold);

		hipDeviceSynchronize();
		hipMemcpy(&markFalgOnHost, markFlagOnDevice, sizeof(bool), hipMemcpyDeviceToHost);

		if (markFalgOnHost)
		{
			analysis <<< grid, threads >>>(LabelListOnDevice, ReferenceOnDevice, width, height);
			hipDeviceSynchronize();
			labeling <<< grid, threads >>>(LabelListOnDevice, ReferenceOnDevice, width, height);
		}
		else
		{
			break;
		}
	}

	hipMemcpy(labels, LabelListOnDevice, sizeof(int) * N, hipMemcpyDeviceToHost);

	hipFree(FrameDataOnDevice);
	hipFree(LabelListOnDevice);
	hipFree(ReferenceOnDevice);
}




}

#endif
