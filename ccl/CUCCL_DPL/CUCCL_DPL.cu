#include "hip/hip_runtime.h"
#ifndef CUCCL_DPL_CU
#define CUCCL_DPL_CU

#include <host_defines.h>
#include "CUCCL_DPL.cuh"
#include <>
#include <cmath>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>


namespace CUCCL{

const int BLOCK = 8;

__global__ void init_CCLDPL(int labelOnDevice[], int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int id = x + y * width;

	labelOnDevice[id] = id;
}

__device__ unsigned char DiffDPL(unsigned char d1, unsigned char d2)
{
	return abs(d1 - d2);
}

__global__ void kernelDPL(int I, unsigned char dataOnDevice[], int labelOnDevice[], bool* markFlagOnDevice, int N, int width, int height, int threshold)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int id = x + y * width;
	int H = N / width;
	int S, E, step;
	switch (I)
	{
	case 0:
		if (id >= width)
			return;
		S = id;
		E = width * (H - 1) + id;
		step = width;
		break;
	case 1:
		if (id >= H)
			return;
		S = id * width;
		E = S + width - 1;
		step = 1;
		break;
	case 2:
		if (id >= width) return;
		S = width * (H - 1) + id;
		E = id;
		step = - width;
		break;
	case 3:
		if (id >= H) return;
		S = (id + 1) * width - 1;
		E = id * width;
		step = -1;
		break;
	}

	int label = labelOnDevice[S];
	for (int n = S + step; n != E + step; n += step)
	{
		if (DiffDPL(dataOnDevice[n], dataOnDevice[n - step]) <= threshold && label < labelOnDevice[n])
		{
			labelOnDevice[n] = label;
			*markFlagOnDevice = true;
		}
		else label = labelOnDevice[n];
	}
}

__global__ void kernelDPL8(int I, unsigned char dataOnDevice[], int labelOnDevice[], bool* markFlagOnDevice, int N, int width, int height, int threshold)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int id = x + y * width;
	int H = N / width;
	int S, E1, E2, step;
	switch (I)
	{
	case 0:
		if (id >= width + H - 1) return;
		if (id < width) S = id;
		else S = (id - width + 1) * width;
		E1 = width - 1; // % W
		E2 = H - 1; // / W
		step = width + 1;
		break;
	case 1:
		if (id >= width + H - 1) return;
		if (id < width) S = width * (H - 1) + id;
		else S = (id - width + 1) * width;
		E1 = width - 1; // % W
		E2 = 0; // / W
		step = -width + 1;
		break;
	case 2:
		if (id >= width + H - 1) return;
		if (id < width) S = width * (H - 1) + id;
		else S = (id - width) * width + width - 1;
		E1 = 0; // % W
		E2 = 0; // / W
		step = -(width + 1);
		break;
	case 3:
		if (id >= width + H - 1) return;
		if (id < width) S = id;
		else S = (id - width + 1) * width + width - 1;
		E1 = 0; // % W
		E2 = H - 1; // / W
		step = width - 1;
		break;
	}

	if (E1 == S % width || E2 == S / width)
		return;
	int label = labelOnDevice[S];
	for (int n = S + step;; n += step)
	{
		if (DiffDPL(dataOnDevice[n], dataOnDevice[n - step]) <= threshold && label < labelOnDevice[n])
		{
			labelOnDevice[n] = label;
			*markFlagOnDevice = true;
		}
		else label = labelOnDevice[n];
		if (E1 == n % width || E2 == n / width)
			break;
	}
}

void CCLDPLGPU::CudaCCL(unsigned char* frame, int* labels, int width, int height, int degreeOfConnectivity, unsigned char threshold)
{
	auto N = width * height;

	hipMalloc(reinterpret_cast<void**>(&LabelListOnDevice), sizeof(int) * N);
	hipMalloc(reinterpret_cast<void**>(&FrameDataOnDevice), sizeof(unsigned char) * N);

	hipMemcpy(FrameDataOnDevice, frame, sizeof(unsigned char) * N, hipMemcpyHostToDevice);

	bool* markFlagOnDevice;
	hipMalloc(reinterpret_cast<void**>(&markFlagOnDevice), sizeof(bool));

	dim3 grid((width + BLOCK - 1) / BLOCK, (height + BLOCK - 1) / BLOCK);
	dim3 threads(BLOCK, BLOCK);

	init_CCLDPL<<<grid, threads >>>(LabelListOnDevice, width, height);

	auto initLabel = static_cast<int*>(malloc(sizeof(int) * width * height));

	hipMemcpy(initLabel, LabelListOnDevice, sizeof(int) * width * height, hipMemcpyDeviceToHost);
	std::cout << "Init labels:" << std::endl;
	for (auto i = 0; i < height; ++i)
	{
		for (auto j = 0; j < width; ++j)
		{
			std::cout << std::setw(3) << initLabel[i * width + j] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
	free(initLabel);

	while (true)
	{
		auto markFalgOnHost = false;
		hipMemcpy(markFlagOnDevice, &markFalgOnHost, sizeof(bool), hipMemcpyHostToDevice);

		for (int i = 0; i < 4; i++)
		{
			kernelDPL<<< grid, threads >>>(i, FrameDataOnDevice, LabelListOnDevice, markFlagOnDevice, N, width, height, threshold);
			if (degreeOfConnectivity == 8)
			{
				kernelDPL<<< grid, threads>>>(i, FrameDataOnDevice, LabelListOnDevice, markFlagOnDevice, N, width, height, threshold);
			}
		}
		hipMemcpy(&markFalgOnHost, markFlagOnDevice, sizeof(bool), hipMemcpyDeviceToHost);

		if (markFalgOnHost)
		{
			hipDeviceSynchronize();
		}
		else
		{
			break;
		}
	}

	hipMemcpy(labels, LabelListOnDevice, sizeof(int) * N, hipMemcpyDeviceToHost);

	hipFree(FrameDataOnDevice);
	hipFree(LabelListOnDevice);
}


}

#endif